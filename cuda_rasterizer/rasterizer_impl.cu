#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include "rasterizer_impl.h"
#include <iostream>
#include <fstream>
#include <algorithm>
#include <numeric>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include <hipcub/hipcub.hpp>
#include <cub/device/device_radix_sort.cuh>
#define GLM_FORCE_CUDA
#include <glm/glm.hpp>

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
namespace cg = cooperative_groups;

#include "auxiliary.h"
#include "forward.h"
#include "backward.h"

// Helper function to find the next-highest bit of the MSB
// on the CPU.
uint32_t getHigherMsb(uint32_t n)
{
	uint32_t msb = sizeof(n) * 4;
	uint32_t step = msb;
	while (step > 1)
	{
		step /= 2;
		if (n >> msb)
			msb += step;
		else
			msb -= step;
	}
	if (n >> msb)
		msb++;
	return msb;
}

// Wrapper method to call auxiliary coarse frustum containment test.
// Mark all Gaussians that pass it.
__global__ void checkFrustum(int P,
	const float* orig_points,
	const float* viewmatrix,
	const float* projmatrix,
	bool* present)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P)
		return;

	float3 p_view;
	present[idx] = in_frustum(idx, orig_points, viewmatrix, projmatrix, false, p_view);
}

// Generates one key/value pair for all Gaussian / tile overlaps. 
// Run once per Gaussian (1:N mapping).
__global__ void duplicateWithKeys(
	int P,
	const float2* points_xy,
	const float* depths,
	const uint32_t* offsets,
	uint64_t* gaussian_keys_unsorted,
	uint32_t* gaussian_values_unsorted,
	int* radii,
	dim3 grid)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P)
		return;

	// Generate no key/value pair for invisible Gaussians
	if (radii[idx] > 0)
	{
		// Find this Gaussian's offset in buffer for writing keys/values.
		uint32_t off = (idx == 0) ? 0 : offsets[idx - 1];
		uint2 rect_min, rect_max;

		getRect(points_xy[idx], radii[idx], rect_min, rect_max, grid);

		// For each tile that the bounding rect overlaps, emit a 
		// key/value pair. The key is |  tile ID  |      depth      |,
		// and the value is the ID of the Gaussian. Sorting the values 
		// with this key yields Gaussian IDs in a list, such that they
		// are first sorted by tile and then by depth. 
		for (int y = rect_min.y; y < rect_max.y; y++)
		{
			for (int x = rect_min.x; x < rect_max.x; x++)
			{
				uint64_t key = y * grid.x + x;
				key <<= 32;
				key |= *((uint32_t*)&depths[idx]);
				gaussian_keys_unsorted[off] = key;
				gaussian_values_unsorted[off] = idx;
				off++;
			}
		}
	}
}

// Check keys to see if it is at the start/end of one tile's range in 
// the full sorted list. If yes, write start/end of this tile. 
// Run once per instanced (duplicated) Gaussian ID.
__global__ void identifyTileRanges(int L, uint64_t* point_list_keys, uint2* ranges)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= L)
		return;

	// Read tile ID from key. Update start/end of tile range if at limit.
	uint64_t key = point_list_keys[idx];
	uint32_t currtile = key >> 32;
	if (idx == 0)
		ranges[currtile].x = 0;
	else
	{
		uint32_t prevtile = point_list_keys[idx - 1] >> 32;
		if (currtile != prevtile)
		{
			ranges[prevtile].y = idx;
			ranges[currtile].x = idx;
		}
	}
	if (idx == L - 1)
		ranges[currtile].y = L;
}

// Mark Gaussians as visible/invisible, based on view frustum testing
void CudaRasterizer::Rasterizer::markVisible(
	int P,
	float* means3D,
	float* viewmatrix,
	float* projmatrix,
	bool* present)
{
	checkFrustum << <(P + 255) / 256, 256 >> > (
		P,
		means3D,
		viewmatrix, projmatrix,
		present);
}

CudaRasterizer::GeometryState CudaRasterizer::GeometryState::fromChunk(char*& chunk, size_t P)
{
	GeometryState geom;
	obtain(chunk, geom.depths, P, 128);
	obtain(chunk, geom.clamped, P * 3, 128);
	obtain(chunk, geom.internal_radii, P, 128);
	obtain(chunk, geom.means2D, P, 128);
	obtain(chunk, geom.cov3D, P * 6, 128);
	obtain(chunk, geom.conic_opacity, P, 128);
	obtain(chunk, geom.rgb, P * 3, 128);
	obtain(chunk, geom.tiles_touched, P, 128);
	hipcub::DeviceScan::InclusiveSum(nullptr, geom.scan_size, geom.tiles_touched, geom.tiles_touched, P);
	obtain(chunk, geom.scanning_space, geom.scan_size, 128);
	obtain(chunk, geom.point_offsets, P, 128);
	return geom;
}

CudaRasterizer::ImageState CudaRasterizer::ImageState::fromChunk(char*& chunk, size_t N)
{
	ImageState img;
	obtain(chunk, img.accum_alpha, N, 128);
	obtain(chunk, img.n_contrib, N, 128);
	obtain(chunk, img.ranges, N, 128);
	return img;
}

CudaRasterizer::BinningState CudaRasterizer::BinningState::fromChunk(char*& chunk, size_t P)
{
	BinningState binning;
	obtain(chunk, binning.point_list, P, 128);
	obtain(chunk, binning.point_list_unsorted, P, 128);
	obtain(chunk, binning.point_list_keys, P, 128);
	obtain(chunk, binning.point_list_keys_unsorted, P, 128);
	hipcub::DeviceRadixSort::SortPairs(
		nullptr, binning.sorting_size,
		binning.point_list_keys_unsorted, binning.point_list_keys,
		binning.point_list_unsorted, binning.point_list, P);
	obtain(chunk, binning.list_sorting_space, binning.sorting_size, 128);
	return binning;
}

// Forward rendering procedure for differentiable rasterization
// of Gaussians.
int CudaRasterizer::Rasterizer::forward(
	std::function<char* (size_t)> geometryBuffer,			// 3个缓冲区函数
	std::function<char* (size_t)> binningBuffer,
	std::function<char* (size_t)> imageBuffer,
	const int P, int D, int M,								// 3D高斯的数量，球谐函数的阶数，球谐系数的数量
	const float* background,								// 背景颜色
	const int width, int height,							// 图像的宽和高
	const float* means3D,									// 3D高斯的位置均值
	const float* shs,										// 球谐系数
	const float* colors_precomp,							// 预计算的高斯颜色
	const float* opacities,									// 3D高斯的不透明度
	const float* scales,									// 3D高斯的缩放
	const float scale_modifier,								// 3D高斯的缩放修正
	const float* rotations,									// 3D高斯的旋转
	const float* cov3D_precomp,								// 预计算的3D高斯的协方差矩阵
	const float* viewmatrix,								// 视图矩阵T_CW
	const float* projmatrix,								// 经过视图矩阵和投影矩阵相乘后得到的最终投影变换矩阵T_IW
	const float* cam_pos,									// 相机位置
	const float tan_fovx, float tan_fovy,					// 相机水平视场角和垂直视场角
	const bool prefiltered,									// 是否预过滤
	float* out_color,										// 输出的图像颜色
	float* out_depth,										// 输出的深度图
	float* out_opacity,										// 输出的不透明度图（曝光）
	int* radii,												// 2D高斯椭圆的半径
	int* n_touched,											// 每个3D高斯触及到的像素数量
	bool debug)												// 是否调试
{
	const float focal_y = height / (2.0f * tan_fovy);
	const float focal_x = width / (2.0f * tan_fovx);

	// 初始化一些缓冲区
	size_t chunk_size = required<GeometryState>(P);
	char* chunkptr = geometryBuffer(chunk_size);
	GeometryState geomState = GeometryState::fromChunk(chunkptr, P);    // 读取3D高斯的几何信息

	if (radii == nullptr)
	{
		radii = geomState.internal_radii;
	}

	dim3 tile_grid((width + BLOCK_X - 1) / BLOCK_X, (height + BLOCK_Y - 1) / BLOCK_Y, 1);	// x和y方向上栅格的个数
	dim3 block(BLOCK_X, BLOCK_Y, 1);														// 每个栅格的像素大小(线程数)

	// Dynamically resize image-based auxiliary buffers during training
	size_t img_chunk_size = required<ImageState>(width * height);
	char* img_chunkptr = imageBuffer(img_chunk_size);
	ImageState imgState = ImageState::fromChunk(img_chunkptr, width * height);	// 读取图像信息

	if (NUM_CHANNELS != 3 && colors_precomp == nullptr)
	{
		throw std::runtime_error("For non-RGB, provide precomputed Gaussian colors!");
	}

	// 渲染前的预处理
	// Run preprocessing per-Gaussian (transformation, bounding, conversion of SHs to RGB)
	CHECK_CUDA(FORWARD::preprocess(
		P, D, M,
		means3D,
		(glm::vec3*)scales,
		scale_modifier,
		(glm::vec4*)rotations,
		opacities,
		shs,
		geomState.clamped,
		cov3D_precomp,
		colors_precomp,
		viewmatrix, projmatrix,
		(glm::vec3*)cam_pos,
		width, height,
		focal_x, focal_y,
		tan_fovx, tan_fovy,
		radii,
		geomState.means2D,
		geomState.depths,
		geomState.cov3D,
		geomState.rgb,
		geomState.conic_opacity,
		tile_grid,
		geomState.tiles_touched,
		prefiltered
	), debug)

	// Compute prefix sum over full list of touched tile counts by Gaussians
	// E.g., [2, 3, 0, 2, 1] -> [2, 5, 5, 7, 8]
	// 计算每一个栅格被多少个3D高斯触及到geomState.point_offsets
	// 计算出每个高斯对应的keys和values在数组中的起始位置
	CHECK_CUDA(hipcub::DeviceScan::InclusiveSum(geomState.scanning_space, geomState.scan_size, geomState.tiles_touched, geomState.point_offsets, P), debug)

	// Retrieve total number of Gaussian instances to launch and resize aux buffers
	// 计算被渲染的高斯总数
	int num_rendered;
	CHECK_CUDA(hipMemcpy(&num_rendered, geomState.point_offsets + P - 1, sizeof(int), hipMemcpyDeviceToHost), debug);
	// 调整缓冲区大小
	size_t binning_chunk_size = required<BinningState>(num_rendered);
	char* binning_chunkptr = binningBuffer(binning_chunk_size);
	BinningState binningState = BinningState::fromChunk(binning_chunkptr, num_rendered);

	// For each instance to be rendered, produce adequate [ tile | depth ] key 
	// and corresponding dublicated Gaussian indices to be sorted
	// 根据每个3D高斯在相机系下的深度，生成排序所用的keys和values [tile | depth]
	duplicateWithKeys << <(P + 255) / 256, 256 >> > (
		P,
		geomState.means2D,
		geomState.depths,
		geomState.point_offsets,
		binningState.point_list_keys_unsorted,
		binningState.point_list_unsorted,
		radii,
		tile_grid)
	CHECK_CUDA(, debug)

	int bit = getHigherMsb(tile_grid.x * tile_grid.y);

	// Sort complete list of (duplicated) Gaussian indices by keys
	// 进行排序，按keys排序：每个栅格对应的高斯按深度放在一起；value是Gaussian的ID
	CHECK_CUDA(hipcub::DeviceRadixSort::SortPairs(
		binningState.list_sorting_space,
		binningState.sorting_size,
		binningState.point_list_keys_unsorted, binningState.point_list_keys,
		binningState.point_list_unsorted, binningState.point_list,
		num_rendered, 0, 32 + bit), debug)

	CHECK_CUDA(hipMemset(imgState.ranges, 0, tile_grid.x * tile_grid.y * sizeof(uint2)), debug);

	// Identify start and end of per-tile workloads in sorted list
	// 计算每个栅格对应排序过的数组中的哪一部分
	if (num_rendered > 0)
		identifyTileRanges << <(num_rendered + 255) / 256, 256 >> > (
			num_rendered,
			binningState.point_list_keys,
			imgState.ranges);
	CHECK_CUDA(, debug)

	// Let each tile blend its range of Gaussians independently in parallel
	const float* feature_ptr = colors_precomp != nullptr ? colors_precomp : geomState.rgb;  // 3D高斯的颜色
	// 并行渲染图像
	CHECK_CUDA(FORWARD::render(
		tile_grid, block,
		imgState.ranges,
		binningState.point_list,
		width, height,
		geomState.means2D,
		feature_ptr,
		geomState.conic_opacity,
		imgState.accum_alpha,
		imgState.n_contrib,
		background,
		out_color,
		geomState.depths,
		out_depth, 
		out_opacity,
		n_touched
    ), debug)
	// 返回被渲染的高斯总数
	return num_rendered;
}

// Produce necessary gradients for optimization, corresponding
// to forward render pass
void CudaRasterizer::Rasterizer::backward(
	const int P, int D, int M, int R,       	 			// 3D高斯的数量，球谐函数的阶数，球谐系数的数量，被渲染的高斯数量
	const float* background,								// 背景颜色
	const int width, int height,							// 图像的宽和高
	const float* means3D,									// 3D高斯的位置均值
	const float* shs,										// 球谐系数
	const float* colors_precomp,							// 预计算的高斯颜色
	const float* scales,									// 3D高斯的缩放
	const float scale_modifier,								// 3D高斯的缩放修正
	const float* rotations,									// 3D高斯的旋转
	const float* cov3D_precomp,								// 预计算的3D高斯的协方差矩阵
	const float* viewmatrix,								// 视图矩阵T_CW
	const float* projmatrix,								// 经过视图矩阵和投影矩阵相乘后得到的最终投影变换矩阵T_IW
    const float* projmatrix_raw,							// 原始投影矩阵T_IC
    const float* campos,									// 相机位置
	const float tan_fovx, float tan_fovy,					// 相机水平视场角和垂直视场角
	const int* radii,										// 2D高斯椭圆的半径
	char* geom_buffer,										// 高斯几何信息的缓冲区
	char* binning_buffer,									// 光栅化的缓冲区
	char* img_buffer,										// 图像信息的缓冲区
	const float* dL_dpix,									// RGB图像的梯度
	const float* dL_dpix_depth,								// 深度图像的梯度
	float* dL_dmean2D,										// 2D高斯的位置均值的梯度
	float* dL_dconic,										// 2D高斯协方差的梯度
	float* dL_dopacity,										// 3D高斯不透明度的梯度
	float* dL_dcolor,										// 3D高斯颜色的梯度
	float* dL_ddepth,										// 3D高斯在相机系下深度的梯度
	float* dL_dmean3D,										// 3D高斯位置均值的梯度
	float* dL_dcov3D,										// 3D高斯协方差的梯度
	float* dL_dsh,											// sh系数的梯度
	float* dL_dscale,										// 3D高斯缩放的梯度
	float* dL_drot,											// 3D高斯旋转的梯度
	float* dL_dtau,											// 相机位姿的梯度
	bool debug)												// 是否调试
{
	// 读取缓冲区中储存的信息
	GeometryState geomState = GeometryState::fromChunk(geom_buffer, P);
	BinningState binningState = BinningState::fromChunk(binning_buffer, R);
	ImageState imgState = ImageState::fromChunk(img_buffer, width * height);
	
	// 2D高斯的半径
	if (radii == nullptr)
	{
		radii = geomState.internal_radii;
	}

	const float focal_y = height / (2.0f * tan_fovy);	// 相机y方向焦距
	const float focal_x = width / (2.0f * tan_fovx);	// 相机x方向焦距

	const dim3 tile_grid((width + BLOCK_X - 1) / BLOCK_X, (height + BLOCK_Y - 1) / BLOCK_Y, 1); // x和y方向上栅格的个数
	const dim3 block(BLOCK_X, BLOCK_Y, 1);														// 每个栅格的像素大小(线程数)

	// Compute loss gradients w.r.t. 2D mean position, conic matrix,
	// opacity and RGB of Gaussians from per-pixel loss gradients.
	// If we were given precomputed colors and not SHs, use them.
	const float* color_ptr = (colors_precomp != nullptr) ? colors_precomp : geomState.rgb;	// 3D高斯的颜色
    const float* depth_ptr = geomState.depths;												// 3D高斯在相机系下的深度
	/**********************************光栅化渲染的反响传播函数****************************************/
	CHECK_CUDA(BACKWARD::render(
		tile_grid,
		block,
		imgState.ranges,
		binningState.point_list,
		width, height,
		background,
		geomState.means2D,
		geomState.conic_opacity,
		color_ptr,
		depth_ptr,
		imgState.accum_alpha,
		imgState.n_contrib,
		dL_dpix,
		dL_dpix_depth,
		(float3*)dL_dmean2D,
		(float4*)dL_dconic,
		dL_dopacity,
		dL_dcolor,
		dL_ddepth
    ), debug)

	// Take care of the rest of preprocessing. Was the precomputed covariance
	// given to us or a scales/rot pair? If precomputed, pass that. If not,
	// use the one we computed ourselves.
	const float* cov3D_ptr = (cov3D_precomp != nullptr) ? cov3D_precomp : geomState.cov3D;	// 3D高斯的协方差
	/**********************************预处理的反响传播函数****************************************/
	CHECK_CUDA(BACKWARD::preprocess(P, D, M,
		(float3*)means3D,
		radii,
		shs,
		geomState.clamped,
		(glm::vec3*)scales,
		(glm::vec4*)rotations,
		scale_modifier,
		cov3D_ptr,
		viewmatrix,
		projmatrix,
        projmatrix_raw,
		focal_x, focal_y,
		tan_fovx, tan_fovy,
		(glm::vec3*)campos,
		(float3*)dL_dmean2D,
		dL_dconic,
		(glm::vec3*)dL_dmean3D,
		dL_dcolor,
		dL_ddepth,
		dL_dcov3D,
		dL_dsh,
		(glm::vec3*)dL_dscale,
		(glm::vec4*)dL_drot,
		dL_dtau), debug)
}