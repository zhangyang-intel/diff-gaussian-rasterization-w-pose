#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include "forward.h"
#include "auxiliary.h"
#include "hip/hip_vector_types.h"
#include "math.h"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
namespace cg = cooperative_groups;

// Forward method for converting the input spherical harmonics
// coefficients of each Gaussian to a simple RGB color.
__device__ glm::vec3 computeColorFromSH(int idx, int deg, int max_coeffs, const glm::vec3* means, glm::vec3 campos, const float* shs, bool* clamped)
{
	// The implementation is loosely based on code for 
	// "Differentiable Point-Based Radiance Fields for 
	// Efficient View Synthesis" by Zhang et al. (2022)
	glm::vec3 pos = means[idx];
	glm::vec3 dir = pos - campos;
	dir = dir / glm::length(dir);

	glm::vec3* sh = ((glm::vec3*)shs) + idx * max_coeffs;
	glm::vec3 result = SH_C0 * sh[0];

	if (deg > 0)
	{
		float x = dir.x;
		float y = dir.y;
		float z = dir.z;
		result = result - SH_C1 * y * sh[1] + SH_C1 * z * sh[2] - SH_C1 * x * sh[3];

		if (deg > 1)
		{
			float xx = x * x, yy = y * y, zz = z * z;
			float xy = x * y, yz = y * z, xz = x * z;
			result = result +
				SH_C2[0] * xy * sh[4] +
				SH_C2[1] * yz * sh[5] +
				SH_C2[2] * (2.0f * zz - xx - yy) * sh[6] +
				SH_C2[3] * xz * sh[7] +
				SH_C2[4] * (xx - yy) * sh[8];

			if (deg > 2)
			{
				result = result +
					SH_C3[0] * y * (3.0f * xx - yy) * sh[9] +
					SH_C3[1] * xy * z * sh[10] +
					SH_C3[2] * y * (4.0f * zz - xx - yy) * sh[11] +
					SH_C3[3] * z * (2.0f * zz - 3.0f * xx - 3.0f * yy) * sh[12] +
					SH_C3[4] * x * (4.0f * zz - xx - yy) * sh[13] +
					SH_C3[5] * z * (xx - yy) * sh[14] +
					SH_C3[6] * x * (xx - 3.0f * yy) * sh[15];
			}
		}
	}
	result += 0.5f;

	// RGB colors are clamped to positive values. If values are
	// clamped, we need to keep track of this for the backward pass.
	clamped[3 * idx + 0] = (result.x < 0);
	clamped[3 * idx + 1] = (result.y < 0);
	clamped[3 * idx + 2] = (result.z < 0);
	return glm::max(result, 0.0f);
}

// Forward version of 2D covariance matrix computation
__device__ float3 computeCov2D(const float3& mean, float focal_x, float focal_y, float tan_fovx, float tan_fovy, const float* cov3D, const float* viewmatrix)
{
	// The following models the steps outlined by equations 29
	// and 31 in "EWA Splatting" (Zwicker et al., 2002). 
	// Additionally considers aspect / scaling of viewport.
	// Transposes used to account for row-/column-major conventions.
	// 当前3D高斯的位置均值在相机系下的坐标
	float3 t = transformPoint4x3(mean, viewmatrix);

	// 限制3D高斯在相机视锥体范围内
	const float limx = 1.3f * tan_fovx;
	const float limy = 1.3f * tan_fovy;
	const float txtz = t.x / t.z;
	const float tytz = t.y / t.z;
	t.x = min(limx, max(-limx, txtz)) * t.z;
	t.y = min(limy, max(-limy, tytz)) * t.z;

	// 当前2D高斯的2D协方差矩阵cov Σ′ = J^T * W^T * Σ * W * J
	glm::mat3 J = glm::mat3(
		focal_x / t.z, 0.0f, -(focal_x * t.x) / (t.z * t.z),
		0.0f, focal_y / t.z, -(focal_y * t.y) / (t.z * t.z),
		0, 0, 0);

	glm::mat3 W = glm::mat3(
		viewmatrix[0], viewmatrix[4], viewmatrix[8],
		viewmatrix[1], viewmatrix[5], viewmatrix[9],
		viewmatrix[2], viewmatrix[6], viewmatrix[10]);

	glm::mat3 T = W * J;

	glm::mat3 Vrk = glm::mat3(
		cov3D[0], cov3D[1], cov3D[2],
		cov3D[1], cov3D[3], cov3D[4],
		cov3D[2], cov3D[4], cov3D[5]);

	glm::mat3 cov = glm::transpose(T) * glm::transpose(Vrk) * T;

	// Apply low-pass filter: every Gaussian should be at least
	// one pixel wide/high. Discard 3rd row and column.
	cov[0][0] += 0.3f;
	cov[1][1] += 0.3f;
	return { float(cov[0][0]), float(cov[0][1]), float(cov[1][1]) };
}

// Forward method for converting scale and rotation properties of each
// Gaussian to a 3D covariance matrix in world space. Also takes care
// of quaternion normalization.
__device__ void computeCov3D(const glm::vec3 scale, float mod, const glm::vec4 rot, float* cov3D)
{
	// Create scaling matrix
	glm::mat3 S = glm::mat3(1.0f);
	S[0][0] = mod * scale.x;
	S[1][1] = mod * scale.y;
	S[2][2] = mod * scale.z;

	// Normalize quaternion to get valid rotation
	glm::vec4 q = rot;// / glm::length(rot);
	float r = q.x;
	float x = q.y;
	float y = q.z;
	float z = q.w;

	// Compute rotation matrix from quaternion
	glm::mat3 R = glm::mat3(
		1.f - 2.f * (y * y + z * z), 2.f * (x * y - r * z), 2.f * (x * z + r * y),
		2.f * (x * y + r * z), 1.f - 2.f * (x * x + z * z), 2.f * (y * z - r * x),
		2.f * (x * z - r * y), 2.f * (y * z + r * x), 1.f - 2.f * (x * x + y * y)
	);

	glm::mat3 M = S * R;

	// Compute 3D world covariance matrix Sigma
	glm::mat3 Sigma = glm::transpose(M) * M;

	// Covariance is symmetric, only store upper right
	cov3D[0] = Sigma[0][0];
	cov3D[1] = Sigma[0][1];
	cov3D[2] = Sigma[0][2];
	cov3D[3] = Sigma[1][1];
	cov3D[4] = Sigma[1][2];
	cov3D[5] = Sigma[2][2];
}

// Perform initial steps for each Gaussian prior to rasterization.
template<int C>
__global__ void preprocessCUDA(int P, int D, int M,		// 3D高斯的数量，球谐系数的数量，球谐系数的最大度数
	const float* orig_points,							// 3D高斯的位置均值
	const glm::vec3* scales,							// 3D高斯的缩放
	const float scale_modifier,							// 3D高斯的缩放修正
	const glm::vec4* rotations,							// 3D高斯的旋转
	const float* opacities,								// 3D高斯的不透明度
	const float* shs,									// 球谐系数
	bool* clamped,										// 被剔除的3D高斯(RGB颜色为负值)
	const float* cov3D_precomp,							// 预计算的3D高斯协方差矩阵
	const float* colors_precomp,						// 预计算的3D高斯颜色
	const float* viewmatrix,							// 视图矩阵T_CW
	const float* projmatrix,							// 经过视图矩阵和投影矩阵相乘后得到的最终投影变换矩阵T_IW
	const glm::vec3* cam_pos,							// 相机位置
	const int W, int H,									// 图像宽度和高度
	const float tan_fovx, float tan_fovy,				// 相机水平和垂直视场角的正切值
	const float focal_x, float focal_y,					// 焦距
	int* radii,											// 2D高斯椭圆的半径
	float2* points_xy_image,							// 2D高斯的位置均值
	float* depths,										// 3D高斯的深度（3D高斯中心在相机系下z轴的坐标）
	float* cov3Ds,										// 3D高斯的协方差矩阵
	float* rgb,											// 3D高斯的颜色
	float4* conic_opacity,								// 2D高斯的协方差矩阵和不透明度
	const dim3 grid,									// 栅格大小（x和y方向上的栅格数）
	uint32_t* tiles_touched,							// 每个3D高斯触及到的栅格数量
	bool prefiltered)									// 是否预过滤
{
	// 获取当前线程的全局索引
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P)
		return;

	// Initialize radius and touched tiles to 0. If this isn't changed,
	// this Gaussian will not be processed further.
	// 初始化2D高斯椭圆的半径和当前高斯触及到的栅格数
	radii[idx] = 0;
	tiles_touched[idx] = 0;

	// Perform near culling, quit if outside.
	// 剔除距离相机太近的高斯，计算当前3D高斯在相机系下的坐标p_view
	float3 p_view;
	if (!in_frustum(idx, orig_points, viewmatrix, projmatrix, prefiltered, p_view))
		return;

	// Transform point by projecting
	// 计算当前3D高斯位置均值在图像系下的坐标
	float3 p_orig = { orig_points[3 * idx], orig_points[3 * idx + 1], orig_points[3 * idx + 2] };
	float4 p_hom = transformPoint4x4(p_orig, projmatrix);
	float p_w = 1.0f / (p_hom.w + 0.0000001f);
	float3 p_proj = { p_hom.x * p_w, p_hom.y * p_w, p_hom.z * p_w };  // (x,y,z) -> (x,y,z,w) -> (x/w,y/w,z/w)

	// If 3D covariance matrix is precomputed, use it, otherwise compute
	// from scaling and rotation parameters. 
	// 计算或使用预计算当前的3D高斯的协方差矩阵
	const float* cov3D;
	if (cov3D_precomp != nullptr)
	{
		cov3D = cov3D_precomp + idx * 6;  // 对称矩阵，只存储上三角6个元素
	}
	else
	{
		computeCov3D(scales[idx], scale_modifier, rotations[idx], cov3Ds + idx * 6);
		cov3D = cov3Ds + idx * 6;
	}

	// Compute 2D screen-space covariance matrix
	// 计算当前2D高斯的协方差矩阵
	float3 cov = computeCov2D(p_orig, focal_x, focal_y, tan_fovx, tan_fovy, cov3D, viewmatrix);

	// Invert covariance (EWA algorithm)
	// 计算当前2D高斯的协方差矩阵的逆矩阵conic Σ′^(-1)
	float det = (cov.x * cov.z - cov.y * cov.y);
	if (det == 0.0f)
		return;
	float det_inv = 1.f / det;
	float3 conic = { cov.z * det_inv, -cov.y * det_inv, cov.x * det_inv };

	// Compute extent in screen space (by finding eigenvalues of
	// 2D covariance matrix). Use extent to compute a bounding rectangle
	// of screen-space tiles that this Gaussian overlaps with. Quit if
	// rectangle covers 0 tiles. 
	// 计算当前2D高斯协方差矩阵的特征值
	float mid = 0.5f * (cov.x + cov.z);
	float lambda1 = mid + sqrt(max(0.1f, mid * mid - det));
	float lambda2 = mid - sqrt(max(0.1f, mid * mid - det));
	// 计算当前2D高斯椭圆(圆)的半径
	float my_radius = ceil(3.f * sqrt(max(lambda1, lambda2)));
	// 计算当前2D高斯在像素系下的像素坐标
	float2 point_image = { ndc2Pix(p_proj.x, W), ndc2Pix(p_proj.y, H) };
	// 计算当前2D高斯外接矩形相交的栅格范围
	uint2 rect_min, rect_max;
	getRect(point_image, my_radius, rect_min, rect_max, grid);
	if ((rect_max.x - rect_min.x) * (rect_max.y - rect_min.y) == 0)
		return;

	// If colors have been precomputed, use them, otherwise convert
	// spherical harmonics coefficients to RGB color.
	// 使用球谐函数的系数计算当前3D高斯的RGB颜色存储到缓冲区
	if (colors_precomp == nullptr)
	{
		glm::vec3 result = computeColorFromSH(idx, D, M, (glm::vec3*)orig_points, *cam_pos, shs, clamped);
		rgb[idx * C + 0] = result.x;
		rgb[idx * C + 1] = result.y;
		rgb[idx * C + 2] = result.z;
	}

	// Store some useful helper data for the next steps.
	// 存储当前高斯的几何信息到缓冲区
	depths[idx] = p_view.z;														// 当前3D高斯的深度
	radii[idx] = my_radius;														// 当前2D高斯的半径
	points_xy_image[idx] = point_image;											// 当前2D高斯的位置均值
	// Inverse 2D covariance and opacity neatly pack into one float4
	conic_opacity[idx] = { conic.x, conic.y, conic.z, opacities[idx] };			// 当前2D高斯的协方差矩阵和不透明度打包
	tiles_touched[idx] = (rect_max.y - rect_min.y) * (rect_max.x - rect_min.x);	// 当前3D高斯触及到的栅格数量
}

// Main rasterization method. Collaboratively works on one tile per
// block, each thread treats one pixel. Alternates between fetching 
// and rasterizing data.
template <uint32_t CHANNELS>
__global__ void __launch_bounds__(BLOCK_X * BLOCK_Y)
renderCUDA(
	const uint2* __restrict__ ranges,						// 包含每个栅格排序的高斯的起始和结束位置索引的数组
	const uint32_t* __restrict__ point_list,				// 包含所有高斯的索引的数组
	int W, int H,											// 图像宽度和高度
	const float2* __restrict__ points_xy_image,				// 所有2D高斯的位置均值
	const float* __restrict__ features,						// 所有3D高斯的颜色
	const float4* __restrict__ conic_opacity,				// 所有2D高斯的协方差矩阵和不透明度
	float* __restrict__ final_T,							// 存储每个像素最终透光率的数组
	uint32_t* __restrict__ n_contrib,						// 存储对每个像素颜色有贡献的高斯计数
	const float* __restrict__ bg_color,						// 背景颜色
	float* __restrict__ out_color,							// 输出的图像颜色
	const float* __restrict__ depth,						// 所有3D高斯在相机系下的深度
	float* __restrict__ out_depth, 							// 输出的深度图	
	float* __restrict__ out_opacity,						// 输出的不透明度图（曝光）
	int * __restrict__ n_touched)							// 每个3D高斯触及到的像素数
{
	// Identify current tile and associated min/max pixel range.
	// 读取当前栅格block和相关的最小/最大像素范围pix_min～pix_max
	auto block = cg::this_thread_block();
    uint32_t horizontal_blocks = (W + BLOCK_X - 1) / BLOCK_X;
	// uint32_t horizontal_blocks = gridDim.x; # TODO Maybe it's different?
	uint2 pix_min = { block.group_index().x * BLOCK_X, block.group_index().y * BLOCK_Y };
	uint2 pix_max = { min(pix_min.x + BLOCK_X, W), min(pix_min.y + BLOCK_Y , H) };

	// 当前线程像素点的像素坐标pix和像素ID
	uint2 pix = { pix_min.x + block.thread_index().x, pix_min.y + block.thread_index().y };
	uint32_t pix_id = W * pix.y + pix.x;
	float2 pixf = { (float)pix.x, (float)pix.y };

	// Check if this thread is associated with a valid pixel or outside.
	// 判断当前像素点是否在图像范围内
	bool inside = pix.x < W&& pix.y < H;
	// Done threads can help with fetching, but don't rasterize
	bool done = !inside;

	// Load start/end range of IDs to process in bit sorted list.
	// 读取当前栅格中的高斯的起始和结束位置索引range，并计算需要处理的高斯数量toDo，对其进行分批次处理rounds
	uint2 range = ranges[block.group_index().y * horizontal_blocks + block.group_index().x];
	const int rounds = ((range.y - range.x + BLOCK_SIZE - 1) / BLOCK_SIZE);
	int toDo = range.y - range.x;

	// Allocate storage for batches of collectively fetched data.
	// 为当前栅格中的每个线程分配共享存储空间
	__shared__ int collected_id[BLOCK_SIZE];				// 当前栅格中所有高斯的ID
	__shared__ float2 collected_xy[BLOCK_SIZE];				// 当前栅格中所有2D高斯的位置均值
	__shared__ float4 collected_conic_opacity[BLOCK_SIZE];	// 当前栅格中所有2D高斯的协方差矩阵和不透明度
	__shared__ float collected_depth[BLOCK_SIZE];			// 当前栅格中所有3D高斯的深度

	// Initialize helper variables
	// 初始化渲染相关变量
	float T = 1.0f;					// 当前高斯的透光率
	uint32_t contributor = 0;		// 对当前像素有贡献的高斯的ID
	uint32_t last_contributor = 0;	// 对当前像素有贡献的最后一个高斯的ID
	float C[CHANNELS] = { 0 };		// 当前像素的颜色
	float D = 0.0f;					// 当前像素的深度

	// Iterate over batches until all done or range is complete
	// 逐批次处理高斯，每批次处理256个高斯
	for (int i = 0; i < rounds; i++, toDo -= BLOCK_SIZE)
	{
		// End if entire block votes that it is done rasterizing
		// 线程块内是否所有线程都已经完成渲染，退出循环
		int num_done = __syncthreads_count(done);
		if (num_done == BLOCK_SIZE)
			break;

		// Collectively fetch per-Gaussian data from global to shared
		// 读取当前栅格中的高斯的数据到共享内存中
		int progress = i * BLOCK_SIZE + block.thread_rank();	// 当前线程处理的高斯索引
		if (range.x + progress < range.y)
		{
			int coll_id = point_list[range.x + progress];
			collected_id[block.thread_rank()] = coll_id;							// 当前高斯的ID
			collected_xy[block.thread_rank()] = points_xy_image[coll_id];			// 当前2D高斯的位置均值
			collected_conic_opacity[block.thread_rank()] = conic_opacity[coll_id];	// 当前2D高斯的协方差矩阵和不透明度
			collected_depth[block.thread_rank()] = depth[coll_id];					// 当前3D高斯在相机系下的深度
		}
		// 等待所有线程读取完毕再继续
		block.sync();

		// Iterate over current batch
		// 遍历当前批次的高斯
		for (int j = 0; !done && j < min(BLOCK_SIZE, toDo); j++)
		{
			// Keep track of current position in range
			// 更新当前线程负责像素贡献高斯的ID
			contributor++;

			// Resample using conic matrix (cf. "Surface 
			// Splatting" by Zwicker et al., 2001)
			// 计算当前2D高斯对于当前像素点的不透明度power：当前2D高斯的位置均值和当前像素位置之间的距离
			float2 xy = collected_xy[j];
			float2 d = { xy.x - pixf.x, xy.y - pixf.y };
			float4 con_o = collected_conic_opacity[j];
			float power = -0.5f * (con_o.x * d.x * d.x + con_o.z * d.y * d.y) - con_o.y * d.x * d.y;
			if (power > 0.0f)
				continue;

			// Eq. (2) from 3D Gaussian splatting paper.
			// Obtain alpha by multiplying with Gaussian opacity
			// and its exponential falloff from mean.
			// Avoid numerical instabilities (see paper appendix). 
			// 计算当前高斯的不透明度alpha
			float alpha = min(0.99f, con_o.w * exp(power));
			if (alpha < 1.0f / 255.0f) {
				continue;
			}
			// 计算当前高斯的透光率T
			float test_T = T * (1 - alpha);
			if (test_T < 0.0001f)
			{
				done = true;
				continue;
			}
			// Eq. (3) from 3D Gaussian splatting paper.
			// 计算当前像素点的颜色C和深度D
			for (int ch = 0; ch < CHANNELS; ch++) {
				C[ch] += features[collected_id[j] * CHANNELS + ch] * alpha * T;	// C = C + ci * αi * T
			}
			D += collected_depth[j] * alpha * T;	// D = D + di * αi * T
			// Keep track of how many pixels touched this Gaussian.
			// 当前高斯的透光率T大于0.5时，当前3D高斯触及到的像素数+1
			if (test_T > 0.5f) {
				atomicAdd(&(n_touched[collected_id[j]]), 1);
			}
			T = test_T;

			// Keep track of last range entry to update this
			// pixel.
			// 更新当前像素最终的贡献高斯的ID
			last_contributor = contributor;
		}
	}

	// All threads that treat valid pixel write out their final
	// rendering data to the frame and auxiliary buffers.
	// 所有线程都处理完毕后，将最终渲染数据写入到输出缓冲区
	if (inside)
	{
		final_T[pix_id] = T;											// 当前像素的透光率
		n_contrib[pix_id] = last_contributor;							// 对当前像素有贡献的高斯数量
		for (int ch = 0; ch < CHANNELS; ch++) {							// 当前像素的颜色
			out_color[ch * H * W + pix_id] = C[ch] + T * bg_color[ch];
		}
		out_depth[pix_id] = D;											// 当前像素的深度
		out_opacity[pix_id] = 1 - T;									// 当前像素的不透明度
	}
}

void FORWARD::render(
	const dim3 grid, dim3 block,
	const uint2* ranges,
	const uint32_t* point_list,
	int W, int H,
	const float2* means2D,
	const float* colors,
	const float4* conic_opacity,
	float* final_T,
	uint32_t* n_contrib,
	const float* bg_color,
	float* out_color,
	const float* depth,
	float* out_depth, 
	float* out_opacity,
	int* n_touched)
{
	renderCUDA<NUM_CHANNELS> << <grid, block >> > (
		ranges,
		point_list,
		W, H,
		means2D,
		colors,
		conic_opacity,
		final_T,
		n_contrib,
		bg_color,
		out_color,
		depth,
		out_depth,
		out_opacity,
		n_touched);
}

void FORWARD::preprocess(int P, int D, int M,
	const float* means3D,
	const glm::vec3* scales,
	const float scale_modifier,
	const glm::vec4* rotations,
	const float* opacities,
	const float* shs,
	bool* clamped,
	const float* cov3D_precomp,
	const float* colors_precomp,
	const float* viewmatrix,
	const float* projmatrix,
	const glm::vec3* cam_pos,
	const int W, int H,
	const float focal_x, float focal_y,
	const float tan_fovx, float tan_fovy,
	int* radii,
	float2* means2D,
	float* depths,
	float* cov3Ds,
	float* rgb,
	float4* conic_opacity,
	const dim3 grid,
	uint32_t* tiles_touched,
	bool prefiltered)
{
	preprocessCUDA<NUM_CHANNELS> << <(P + 255) / 256, 256 >> > (
		P, D, M,
		means3D,
		scales,
		scale_modifier,
		rotations,
		opacities,
		shs,
		clamped,
		cov3D_precomp,
		colors_precomp,
		viewmatrix, 
		projmatrix,
		cam_pos,
		W, H,
		tan_fovx, tan_fovy,
		focal_x, focal_y,
		radii,
		means2D,
		depths,
		cov3Ds,
		rgb,
		conic_opacity,
		grid,
		tiles_touched,
		prefiltered
		);
}